#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
        my_abort(err); }


// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input_a, float *input_b, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = sqrt(input_a[tid]*input_a[tid] + input_b[tid]*input_b[tid]);
}


// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)rand() / RAND_MAX;
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostData_a, float *hostData_b, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    // Allocate data on GPU memory
    float *deviceInputData_a = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceInputData_a, dataSize * sizeof(float)));
    float *deviceInputData_b = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceInputData_b, dataSize * sizeof(float)));

    float *deviceOutputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

    // Copy to GPU memory
    CUDA_CHECK(hipMemcpy(deviceInputData_a, hostData_a, dataSize * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceInputData_b, hostData_b, dataSize * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData_a, deviceInputData_b, deviceOutputData);

    // Copy data back to CPU memory
    CUDA_CHECK(hipMemcpy(hostData_a, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(deviceInputData_a));
    CUDA_CHECK(hipFree(deviceInputData_b));
    CUDA_CHECK(hipFree(deviceOutputData));
}

float max_here(float *data, int size)
{
    float max_val = data[0];

    for (int i = 1; i < size; i++)
    {
        if(data[i] > max_val){
            max_val = data[i];
        }
    }

    return max_val;
}
